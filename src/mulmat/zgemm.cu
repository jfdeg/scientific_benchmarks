#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>

#include "gene_bruit_rayleigh_scalaire.c"


int main (void)
{


int M,N;
int kboucle;


FILE *fichier1;

fichier1=fopen("../results/results_cuda_gemm_fp64.dat","w");

hipSetDevice(1);
/* Debut grand boucle */

for (kboucle=1 ; kboucle<31 ;kboucle++)
{

M=100*kboucle;

N=M;

printf(">>>>> Matrix size %dx%d  <<<<<<\n",M,N);

int iboucle,dim;
int param=20;
float *mat_real,*mat_imag;
float charge;

hipblasStatus_t status ;

// Chronometre

  struct timespec tpdeb,tpfin;
  clockid_t clock_id=CLOCK_REALTIME;
  int status2;

  struct timespec tpdeb2,tpfin2;

  float dureeloc = 0.0;


// BLAS

hipblasOperation_t transa,transb;


  /* CUBLAS */

float time1; 
hipEvent_t start1, stop;

hipDoubleComplex cualpha,cubeta;
hipDoubleComplex *h_A,*h_B;
hipDoubleComplex *h_C;
hipDoubleComplex* d_A;
hipDoubleComplex* d_B;
hipDoubleComplex* d_C;

transa=HIPBLAS_OP_N;
transb=HIPBLAS_OP_N;


cualpha.x=1.0;
cualpha.y=0.0;
cubeta.x=0.0;
cubeta.y=0.0;

dim=M;

mat_real=(float*)calloc(M*N,sizeof(float));
mat_imag=(float*)calloc(M*N,sizeof(float));

h_A=(hipDoubleComplex*)calloc(M*N,sizeof(hipDoubleComplex));
h_B=(hipDoubleComplex*)calloc(M*N,sizeof(hipDoubleComplex));
h_C=(hipDoubleComplex*)calloc(M*N,sizeof(hipDoubleComplex));

    /* Initialize CUBLAS */

hipblasHandle_t handle;
status=hipblasCreate(&handle);


 hipMalloc((void**)&d_A, M*N*sizeof(hipDoubleComplex));
 hipMalloc((void**)&d_B, M*N*sizeof(hipDoubleComplex));
 hipMalloc((void**)&d_C, M*N*sizeof(hipDoubleComplex));


for (iboucle=0 ; iboucle<M*N ; iboucle++)
{
gene_bruit_rayleigh_scalaire(param,mat_real+iboucle,mat_imag+iboucle);
}


for(iboucle=0 ; iboucle<N*N ; iboucle++)
{
h_A[iboucle].x=mat_real[iboucle];
h_A[iboucle].y=mat_imag[iboucle];
h_B[iboucle].x=mat_imag[iboucle];
h_B[iboucle].y=mat_real[iboucle];
}

 // remise a zero des chronos
  dureeloc=0.0;

  hipEventCreate(&start1);
  hipEventCreate(&stop);
  status2=clock_gettime(clock_id, &tpdeb);


  // Copie de la matrice dans le GPU

  status = hipblasSetVector(M*N, sizeof(hipDoubleComplex), h_A, 1, d_A, 1);

  status2=clock_gettime(clock_id, &tpdeb2);
  hipDeviceSynchronize();
  hipEventRecord(start1, 0);

  hipblasZgemm(handle,transa,transb,dim,dim,dim,&cualpha,d_A,dim,d_B,dim,&cubeta,d_C,dim);


	hipDeviceSynchronize();
	hipEventRecord( stop, 0);

  status2=clock_gettime(clock_id, &tpfin2);

  // Copie matrice 

   status = hipblasGetVector(M*N, sizeof(hipDoubleComplex), d_C, 1, h_C, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }

  status2=clock_gettime(clock_id, &tpfin);
  dureeloc=(float)(tpfin.tv_sec-tpdeb.tv_sec)+(float)(tpfin.tv_nsec-tpdeb.tv_nsec)*1.e-9;

		hipEventElapsedTime( &time1, start1, stop );
		hipEventDestroy( start1 );
		hipEventDestroy( stop );

  charge=(float)M;
  charge=8*charge*charge*charge;
  charge=charge/(time1/1000);
  printf("compute power = %f GFLOPS\n",charge/1e9);
  fprintf(fichier1,"%5.2f\n",charge/1e9);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

hipblasDestroy ( handle ) ;

free(mat_real);
free(mat_imag);

free(h_A);
free(h_B);
free(h_C);

}

fclose(fichier1);

return 0;
}
