#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include "gene_bruit_rayleigh_scalaire.c"


int main (void)
{


int M,N;
int kboucle;

hipSetDevice(1);

FILE *fichier1;

fichier1=fopen("../results/results_cuda_gemm_fp32.dat","w");

/* Debut grand boucle */

for (kboucle=1 ; kboucle<31 ;kboucle++)
{

M=100*kboucle;
N=M;

printf(">>>>> Matrix size %dx%d  <<<<<<\n",M,N);

int iboucle,dim;
int param=20;
float *mat_real,*mat_imag;
float charge;

hipblasStatus_t status;

// Chronometre

  struct timespec tpdeb,tpfin;
  clockid_t clock_id=CLOCK_REALTIME;
  int status2;

  struct timespec tpdeb2,tpfin2;

  float dureeloc,dureetot;
  dureetot=0.0;

// BLAS

hipblasOperation_t transa,transb;


  /* CUBLAS */

float time1; 
hipEvent_t start1, stop;

hipComplex cualpha,cubeta;
hipComplex *h_A,*h_B;
hipComplex *h_C;
hipComplex* d_A;
hipComplex* d_B;
hipComplex* d_C;

transa=HIPBLAS_OP_N;
transb=HIPBLAS_OP_N;


cualpha.x=1.0;
cualpha.y=0.0;
cubeta.x=0.0;
cubeta.y=0.0;

dim=M;

mat_real=(float*)calloc(M*N,sizeof(float));
mat_imag=(float*)calloc(M*N,sizeof(float));

h_A=(hipComplex*)calloc(M*N,sizeof(hipComplex));
h_B=(hipComplex*)calloc(M*N,sizeof(hipComplex));
h_C=(hipComplex*)calloc(M*N,sizeof(hipComplex));

    /* Initialize CUBLAS */

hipblasHandle_t handle;
status=hipblasCreate(&handle);


 hipMalloc((void**)&d_A, M*N*sizeof(hipComplex));
 hipMalloc((void**)&d_B, M*N*sizeof(hipComplex));
 hipMalloc((void**)&d_C, M*N*sizeof(hipComplex));


for (iboucle=0 ; iboucle<M*N ; iboucle++)
{
gene_bruit_rayleigh_scalaire(param,mat_real+iboucle,mat_imag+iboucle);
}


for(iboucle=0 ; iboucle<N*N ; iboucle++)
{
h_A[iboucle].x=mat_real[iboucle];
h_A[iboucle].y=mat_imag[iboucle];
h_B[iboucle].x=mat_imag[iboucle];
h_B[iboucle].y=mat_real[iboucle];
}


  // Remise a zero et deuxieme chorno



  dureeloc=0.0;
  dureetot=0.0;

  hipEventCreate(&start1);
  hipEventCreate(&stop);
  status2=clock_gettime(clock_id, &tpdeb);

  // Copie de la matrice dans le GPU

  status = hipblasSetVector(M*N, sizeof(hipComplex), h_A, 1, d_A, 1);

  status2=clock_gettime(clock_id, &tpdeb2);
  hipDeviceSynchronize();
  hipEventRecord(start1, 0);

  hipblasCgemm(handle,transa,transb,dim,dim,dim,&cualpha,d_A,dim,d_B,dim,&cubeta,d_C,dim);


	hipDeviceSynchronize();
	hipEventRecord( stop, 0);

  status2=clock_gettime(clock_id, &tpfin2);

   status = hipblasGetVector(M*N, sizeof(hipComplex), d_C, 1, h_C, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
    }

  status2=clock_gettime(clock_id, &tpfin);
  dureeloc=(float)(tpfin.tv_sec-tpdeb.tv_sec)+(float)(tpfin.tv_nsec-tpdeb.tv_nsec)*1.e-9;
  dureetot=dureetot+dureeloc;

		hipEventElapsedTime( &time1, start1, stop );
		hipEventDestroy( start1 );
		hipEventDestroy( stop );

  charge=(float)M;
  charge=8*charge*charge*charge;
  charge=charge/(time1/1000);
  printf("compute power = %f GFLOPS\n",charge/1e9);

  fprintf(fichier1,"%5.2f\n",charge/1e9);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

hipblasDestroy ( handle ) ;

free(mat_real);
free(mat_imag);

free(h_A);
free(h_B);
free(h_C);

}

fclose(fichier1);
return 0;
}
