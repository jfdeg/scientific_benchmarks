#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hipfft/hipfft.h>

#include "gene_bruit_rayleigh_scalaire.c"


int main (int argc, char** argv)
{

hipSetDevice(0);

int NFFT,NPOINTS,NTOT;

NTOT=67108864;
//NTOT=16777216;


/* Chrono */
struct timespec tpdeb,tpfin;
clockid_t clock_id=CLOCK_REALTIME;
int status;
float dureeloc;
float dureetot = 0.0;
hipfftResult_t res;

float time1,time2; 
hipEvent_t start1,start2, stop1,stop2;


FILE *fichier1;
fichier1=fopen("../results/GPU_GFLOPS.dat","w");

/* Generateur de signal aleatoire Gaussien */
float param;
float *scalar_real,*scalar_imag;
float mflops,mflops2;
int i,j;
param=20.0;

//hipSetDevice(1);

hipfftComplex *data,*data_device,*data_device2;
hipMalloc((void**)&data_device, sizeof(hipfftComplex)*NTOT);
hipMalloc((void**)&data_device2, sizeof(hipfftComplex)*NTOT);

data=(hipfftComplex *)calloc(NTOT,sizeof(hipfftComplex));
scalar_real=(float *)calloc(NTOT,sizeof(float));
scalar_imag=(float *)calloc(NTOT,sizeof(float));


 // Debut boucle 
printf("Generation du bruit\n");
for(j = 0 ; j<NTOT ; j++)
{
gene_bruit_rayleigh_scalaire(param,scalar_real+i,scalar_imag+i);
}

for(i = 0 ; i < NTOT ; i++ ) 
	{
        data[i].x = scalar_real[i];
        data[i].y = scalar_imag[i];
        }
for (NPOINTS=2 ; NPOINTS<262144+1 ; NPOINTS=2*NPOINTS)
{
NFFT=NTOT/NPOINTS;
/* Declaration du plan et des donnes */
hipfftHandle plan;


	/*  Creation du plan cufft */

	res=hipfftPlan1d(&plan,NPOINTS, HIPFFT_C2C, NFFT);
        printf("Allocation du plan cuFFT = %i\n",res);
	// Affichage du temps


/*
for (i = 0 ; i<NPOINTS ; i++)
{
fprintf(fichier1,"%20.15e\n",data[i].x);
fprintf(fichier2,"%20.15e\n",data[i].y);
}
*/
		hipEventCreate(&start1);
		hipEventCreate(&start2);
		hipEventCreate(&stop1);
		hipEventCreate(&stop2);
		hipEventSynchronize( start1 );
		hipEventRecord(start1, 0);


	hipMemcpy(data_device, data, sizeof(hipfftComplex)*NPOINTS*NFFT,hipMemcpyHostToDevice);

        status=clock_gettime(clock_id, &tpdeb);
	hipEventSynchronize( start2 );
	hipEventRecord(start2, 0);
	/*----------- Debut calcul de la FFT----------*/

	/* Copie dans la memoire GPU */
	hipDeviceSynchronize();

	res = hipfftExecC2C(plan,data_device,data_device2,HIPFFT_FORWARD);
//	hipfftExecC2C(plan,data_device,data_device,HIPFFT_BACKWARD);

	hipDeviceSynchronize();

	hipEventSynchronize( stop2 );
	hipEventRecord( stop2, 0);

	hipEventSynchronize( stop2 );

	/*---------------------------------------------*/

	hipMemcpy(data, data_device, sizeof(hipfftComplex)*NPOINTS*NFFT,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
        status=clock_gettime(clock_id, &tpfin);

	hipEventSynchronize( stop1 );
	hipEventRecord( stop1, 0);
	hipEventElapsedTime( &time1, start1, stop1 );
	hipEventElapsedTime( &time2, start2, stop2 );
	hipEventDestroy( start1 );
	hipEventDestroy( start2 );
	hipEventDestroy( stop1 );
	hipEventDestroy( stop2 );

        if (status<0)  fprintf(stderr,"Erreur clock_gettime (f:%s n:%d)\n",__FILE__,__LINE__);
        if (status<0)   printf("Erreur CLOCKGETTIME 2");
        dureeloc=(float)(tpfin.tv_sec-tpdeb.tv_sec)+(float)(tpfin.tv_nsec-tpdeb.tv_nsec)*1.e-9;
	dureetot=dureetot+dureeloc;

        printf("Execution de cuFFT = %i\n",res);
	// Affichage du temps

printf("Temps pour %d FFT de %d points complexes = %f ms \n",NFFT,NPOINTS,time2);
printf("Temps GPU avec transfert (timeur CPU) = %f ms \n",dureeloc*1000);
time2=time2/1000;
mflops=NFFT*(5*NPOINTS*(log10(NPOINTS)/log10(2)))/dureeloc;
mflops2=NFFT*(5*NPOINTS*(log10(NPOINTS)/log10(2)))/time2;
printf("Puissance sans transfert = %f Gflops \n",mflops2/1e9);
fprintf(fichier1,"%20.15f\n",mflops2/1e9);


hipfftDestroy(plan);



}
free(data);
free(scalar_real);
free(scalar_imag);
hipFree(data_device);
fclose(fichier1);

return 0;
}
